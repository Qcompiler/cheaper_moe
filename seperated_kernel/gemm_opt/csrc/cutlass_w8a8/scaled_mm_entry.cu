#include "hip/hip_runtime.h"
#include <cudaTypedefs.h>

#include <c10/cuda/CUDAGuard.h>
#include <torch/all.h>

void cutlass_scaled_mm_sm75(torch::Tensor& c, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            c10::optional<torch::Tensor> const& bias);

void cutlass_scaled_mm_sm80(torch::Tensor& c, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            c10::optional<torch::Tensor> const& bias);

void cutlass_scaled_mm_sm89(torch::Tensor& c, torch::Tensor const& a,
                            torch::Tensor const& b,
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            c10::optional<torch::Tensor> const& bias);

#if defined ENABLE_SCALED_MM_C3X && ENABLE_SCALED_MM_C3X
void cutlass_scaled_mm_sm90(torch::Tensor& c, 
                            torch::Tensor const& a,
                            torch::Tensor const& b,
                            int m, int n, int k, 
                            torch::Tensor const& a_scales,
                            torch::Tensor const& b_scales,
                            c10::optional<torch::Tensor> const& bias);
#endif

void cutlass_scaled_mm_azp_sm75(torch::Tensor& c, torch::Tensor const& a,
                                torch::Tensor const& b,
                                torch::Tensor const& a_scales,
                                torch::Tensor const& b_scales,
                                torch::Tensor const& azp_adj,
                                c10::optional<torch::Tensor> const& azp,
                                c10::optional<torch::Tensor> const& bias);

void cutlass_scaled_mm_azp_sm80(torch::Tensor& c, torch::Tensor const& a,
                                torch::Tensor const& b,
                                torch::Tensor const& a_scales,
                                torch::Tensor const& b_scales,
                                torch::Tensor const& azp_adj,
                                c10::optional<torch::Tensor> const& azp,
                                c10::optional<torch::Tensor> const& bias);

void cutlass_scaled_mm_azp_sm89(torch::Tensor& c, torch::Tensor const& a,
                                torch::Tensor const& b,
                                torch::Tensor const& a_scales,
                                torch::Tensor const& b_scales,
                                torch::Tensor const& azp_adj,
                                c10::optional<torch::Tensor> const& azp,
                                c10::optional<torch::Tensor> const& bias);

#if defined CUDA_VERSION && CUDA_VERSION >= 12000
void cutlass_scaled_mm_azp_sm90(torch::Tensor& c, torch::Tensor const& a,
                                torch::Tensor const& b,
                                torch::Tensor const& a_scales,
                                torch::Tensor const& b_scales,
                                torch::Tensor const& azp_adj,
                                c10::optional<torch::Tensor> const& azp,
                                c10::optional<torch::Tensor> const& bias);
#endif

bool cutlass_scaled_mm_supports_fp8(int64_t cuda_device_capability) {
  // CUTLASS FP8 kernels need at least
  //   CUDA 12.0 on SM90 systems (Hopper)
  //   CUDA 12.4 on SM89 systems (Lovelace)

#if defined CUDA_VERSION
  if (cuda_device_capability >= 90) {
    return CUDA_VERSION >= 12000;
  } else if (cuda_device_capability >= 89) {
    return CUDA_VERSION >= 12040;
  }
#endif

  return false;
}

int32_t get_sm_version_num() {
  int32_t major_capability, minor_capability;
  hipDeviceGetAttribute(&major_capability, hipDeviceAttributeComputeCapabilityMajor,
                         0);
  hipDeviceGetAttribute(&minor_capability, hipDeviceAttributeComputeCapabilityMinor,
                         0);
  int32_t version_num = major_capability * 10 + minor_capability;
  return version_num;
}



template<int size>
__global__ void FindRowScaleKernel_(int8_t * output, const half * d_in, float * scale, int rows, int cols){

    __shared__ half sdata[size];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x ;
    if (bid > rows)
        return ;
    const  __half *start = d_in + bid * cols;
    int8_t * d_out = output + bid * cols;
    sdata[tid] = __habs(start[tid]); 
    for (int i = tid + size; i < cols; i += size)
        sdata[tid] = __hmax ( __habs(start[i]),  sdata[tid] ); 
    __syncthreads();


    // do reduction in shared mem
    for (unsigned int s= blockDim.x/2; s >= 1; s >>=1 ) {
        if (tid < s) {
            sdata[tid] =  __hmax ( __habs(sdata[tid + s]),  sdata[tid]);
        }
        __syncthreads();
    }


    // write result for this block to global mem
    //if (tid < 32) warpReduce(sdata, tid);

    __syncthreads();
    float tmp = 127.0;
    half quant_scales = __hdiv( sdata[0], __float2half(tmp));
    if (tid == 0){
        scale[bid] = (__half2float)(quant_scales);
    }
    // quant
    for (int i = tid ; i < cols; i += size)
        d_out[i] =  static_cast<int8_t>(__half2int_rn( __hdiv( start[i], quant_scales ) ))  ; 
    __syncthreads();    

}
void int8quant_(int rows, int cols, const half * src, int8_t *output, 
        float *scale, hipStream_t stream){


    dim3 block(256);
    dim3 grid(rows, 1);
    FindRowScaleKernel_<256><<<grid, block, 1024, stream>>>(
                output,
                src, scale,
                rows, cols);

};
torch::Tensor cutlass_scaled_mm( int batch, int seq_len,  int N, int K,
                       torch::Tensor const& a,
                       torch::Tensor const& b, torch::Tensor const& a_scales,
                       torch::Tensor const& b_scales,
                       c10::optional<torch::Tensor> const& bias) {
  // Checks for conformality

  // TORCH_CHECK(a_scales.numel() == 1 || a_scales.numel() == a.size(0));
  // TORCH_CHECK(b_scales.numel() == 1 || b_scales.numel() == b.size(1));

  // Check for strides and alignment
  // TORCH_CHECK(a.stride(1) == 1 );  // Row-major
  // TORCH_CHECK(b.stride(0) == 1);                      // Column-major
  // TORCH_CHECK(b.stride(1) % 16 == 0);  // 16 Byte Alignment
  TORCH_CHECK(a_scales.is_contiguous() && b_scales.is_contiguous());

  if (bias) {
    TORCH_CHECK(bias->numel() == b.size(1) && bias->is_contiguous() &&
                bias->dim() == 1);
  }

  at::cuda::OptionalCUDAGuard const device_guard(device_of(a));
  int32_t version_num = get_sm_version_num();
  // Hopper
  auto options = torch::TensorOptions().dtype(torch::kFloat16).device(a.device());  
  auto c = torch::zeros(
        {batch, seq_len,  N }, options);

  int M = batch * seq_len;

  auto options_i8 = torch::TensorOptions().dtype(torch::kInt8).device(a.device());
  auto quant_out = torch::zeros(
      { M, K }, options_i8);
  int8_t* int8_out = reinterpret_cast<int8_t *>(quant_out.data_ptr<int8_t>());
  float* scale_a = reinterpret_cast<float *>(a_scales.data_ptr<float>());
  const half * A = reinterpret_cast<half const*>(a.data_ptr<at::Half>());
  const hipStream_t stream = at::cuda::getCurrentCUDAStream(); 
  int8quant_(M, K, A, int8_out, scale_a, stream);
  if (version_num >= 90) {
    cutlass_scaled_mm_sm90(c, quant_out, b, M, N, K, a_scales, b_scales, bias);
    
  }

  return  c;

}



torch::Tensor cutlass_scaled_mm_fp8( int batch, int seq_len,  int N, int K,
                       torch::Tensor const& a,
                       torch::Tensor const& b, torch::Tensor const& a_scales,
                       torch::Tensor const& b_scales,
                       c10::optional<torch::Tensor> const& bias, int dim ) {

  TORCH_CHECK(a_scales.is_contiguous() && b_scales.is_contiguous());

  if (bias) {
    TORCH_CHECK(bias->numel() == b.size(1) && bias->is_contiguous() &&
                bias->dim() == 1);
  }

  at::cuda::OptionalCUDAGuard const device_guard(device_of(a));
  int32_t version_num = get_sm_version_num();
  // Hopper
  auto options = torch::TensorOptions().dtype(torch::kFloat16).device(a.device());  
  torch::Tensor c;

  if (dim == 3)
    c = torch::zeros(
        {batch, seq_len,  N }, options);

  if (dim == 2)
    c = torch::zeros(
        {batch * seq_len,  N }, options); 

  int M = batch * seq_len;

  if (version_num >= 90) {
    cutlass_scaled_mm_sm90(c, a, b, M, N, K, a_scales, b_scales, bias);
    
  }

  return  c;

}

void cutlass_scaled_mm_azp(torch::Tensor& c, torch::Tensor const& a,
                           torch::Tensor const& b,
                           torch::Tensor const& a_scales,
                           torch::Tensor const& b_scales,
                           torch::Tensor const& azp_adj,
                           c10::optional<torch::Tensor> const& azp,
                           c10::optional<torch::Tensor> const& bias) {
  // Checks for conformality
  TORCH_CHECK(a.dim() == 2 && b.dim() == 2 && c.dim() == 2);
  TORCH_CHECK(c.size(0) == a.size(0) && a.size(1) == b.size(0) &&
              b.size(1) == c.size(1));
  TORCH_CHECK(a_scales.numel() == 1 || a_scales.numel() == a.size(0));
  TORCH_CHECK(b_scales.numel() == 1 || b_scales.numel() == b.size(1));

  // Check for strides and alignment
  TORCH_CHECK(a.stride(1) == 1 && c.stride(1) == 1);  // Row-major
  TORCH_CHECK(b.stride(0) == 1);                      // Column-major
  TORCH_CHECK(c.stride(0) % 16 == 0 &&
              b.stride(1) % 16 == 0);  // 16 Byte Alignment
  TORCH_CHECK(a_scales.is_contiguous() && b_scales.is_contiguous());

  // bias, azp, azp_adj are all 1d
  // bias and azp_adj have n elements, azp has m elements
  if (bias) {
    TORCH_CHECK(bias->numel() == b.size(1) && bias->is_contiguous());
  }
  if (azp) {
    TORCH_CHECK(azp->numel() == a.size(0) && azp->is_contiguous());
  }
  TORCH_CHECK(azp_adj.numel() == b.size(1) && azp_adj.is_contiguous());

  // azp & bias types
  TORCH_CHECK(azp_adj.dtype() == torch::kInt32);
  TORCH_CHECK(!azp || azp->dtype() == torch::kInt32);
  TORCH_CHECK(!bias || bias->dtype() == c.dtype(),
              "currently bias dtype must match output dtype ", c.dtype());

  at::cuda::OptionalCUDAGuard const device_guard(device_of(a));

  int32_t version_num = get_sm_version_num();

#if defined ENABLE_SCALED_MM_C3X && ENABLE_SCALED_MM_C3X
  if (version_num >= 90) {
    cutlass_scaled_mm_azp_sm90(c, a, b, a_scales, b_scales, azp_adj, azp, bias);
    return;
  }
#endif

#if defined ENABLE_SCALED_MM_C2X && ENABLE_SCALED_MM_C2X
  if (version_num == 89) {
    // Ada Lovelace
    cutlass_scaled_mm_azp_sm89(c, a, b, a_scales, b_scales, azp_adj, azp, bias);
    return;
  }

  if (version_num >= 80) {
    // Ampere
    cutlass_scaled_mm_azp_sm80(c, a, b, a_scales, b_scales, azp_adj, azp, bias);
    return;
  }

  // Turing
  TORCH_CHECK(version_num >= 75);
  cutlass_scaled_mm_azp_sm75(c, a, b, a_scales, b_scales, azp_adj, azp, bias);
  return;
#endif

  TORCH_CHECK_NOT_IMPLEMENTED(
      false,
      "No compiled cutlass_scaled_mm_azp for a compute capability less than "
      "CUDA device capability: ",
      version_num);
}