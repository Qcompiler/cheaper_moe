#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <hipblas.h>
#include "cutlass/cutlass.h"
#include "utils.cuh"
#include <iostream>

#include "symmetric/gemm/device/gemm_dequant.h"

template<int size>
__global__ void FindRowScaleKernel_(int8_t * output, const half * d_in, half * scale, int rows, int cols){

    __shared__ half sdata[size];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x ;
    if (bid > rows)
        return ;
    const  __half *start = d_in + bid * cols;
    int8_t * d_out = output + bid * cols;
    sdata[tid] = __habs(start[tid]); 
    for (int i = tid + size; i < cols; i += size)
        sdata[tid] = __hmax ( __habs(start[i]),  sdata[tid] ); 
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s= blockDim.x/2; s >= 1; s >>=1 ) {
        if (tid < s) {
            sdata[tid] =  __hmax ( __habs(sdata[tid + s]),  sdata[tid]);
        }
        __syncthreads();
    }

    half  max = sdata[0];
    // write result for this block to global mem
    //if (tid < 32) warpReduce(sdata, tid);

    __syncthreads();

    half quant_scales = __hdiv( max, 127.0);
    if (tid == 0){
        scale[bid] = quant_scales;
    }
    // quant
    for (int i = tid ; i < cols; i += size)
        d_out[i] =  static_cast<int8_t>(__half2int_rn( __hdiv( start[i], quant_scales ) ))  ; 
    __syncthreads();    

}

void int8quant_(int rows, int cols, 
        const cutlass::half_t  * src, int8_t *output, 
        cutlass::half_t  *scale){


    dim3 block(256);
    dim3 grid(rows, 1);
    FindRowScaleKernel_<256><<<grid, block, 1024>>>(
                output,
                (half*) src, 
                (half*) scale,
                rows, cols);

}

void gemmfp16_(
    const half * mat1,
    const half * mat2, half *mat3, int m, int n, int k, hipblasHandle_t handle) {
 

  static float _beta = 0.0;
  static  float _alpha = 1.0;

  auto beta_ptr = (void*)&_beta;
  auto alpha_ptr = (void*)&_alpha;

  auto input_ptr = (void*)mat3;
  auto mat1_ptr = (void*)mat1;
  auto mat2_ptr = (void*)mat2;

    
  (hipblasGemmEx(
       handle,
      HIPBLAS_OP_T,
      HIPBLAS_OP_N,
      n,
      m,
      k,
      alpha_ptr,
      mat2_ptr,
      HIP_R_16F,
      k,
      mat1_ptr,
      HIP_R_16F,
      k,
      beta_ptr,
      input_ptr,
      HIP_R_16F,
      n,
      HIPBLAS_COMPUTE_32F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP));

  

}
void int8FusedDequantizeCUDA_(const int8_t *A,
                             const int8_t *B,
                             const cutlass::half_t *scale_row,
                             const cutlass::half_t *scale_col,
                             cutlass::half_t *y, cutlass::half_t *D, 
                             int M, int N, int K,
                             char * workspace) {

 
  using Gemm = cutlass::gemm::device::symmetric::GemmDequant<
      int8_t,                          // ElementA
      cutlass::layout::RowMajor,       // LayoutA
      int8_t,                          // ElementB
      cutlass::layout::ColumnMajor,    // LayoutB
      cutlass::half_t,                 // ElementOutput
      cutlass::layout::RowMajor,       // LayoutOutput
      int32_t,                         // ElementAccumulator
      cutlass::arch::OpClassTensorOp,  // tag indicating Tensor Cores
      cutlass::arch::Sm80  // tag indicating target GPU compute architecture
      >;

  Gemm gemmOp;
  //cutlass::Status status = gemmOp(stream);
  using GemmCoord = cutlass::gemm::GemmCoord;

  typename Gemm::Arguments arguments{
      {static_cast<GemmCoord::Index>(M), static_cast<GemmCoord::Index>(N),
       static_cast<GemmCoord::Index>(K)},
      {(const int8_t *)A, K},
      {(const int8_t *)B, K},
      {(const cutlass::half_t *)y, N},
      {(cutlass::half_t *)D, N},
      {(const cutlass::half_t *)scale_col, N},
      {(const cutlass::half_t *)scale_row, M},
      Gemm::ElementC(1)};

//   gemmOp.initialize(arguments, workspace);
    auto status = gemmOp(arguments);
//   gemmOp.run();
 
}

__global__  void FindOutliersAndSetToZeros_kernel_(const int *ind,  half *input, 
        half *outliersOutput, int m, int k, int len){
 

    int tid = threadIdx.x;
 
    int start_col = blockIdx.x ;
 
    if (start_col > len)
        return ;

  
 
 
    int col = ind[start_col];
    half *start = input +  col ;
    half *outliersOutput_ = outliersOutput + start_col;   
 
    for (int i = tid; i < m ; i+=  128  ){
        outliersOutput_[ i * len ] = start[ i * k ] ;
        start[ i * k ] = 0.0;
    }
 
 


}
void ExtractOutliersAndSetToZeros_(int M, int N, const half * A, half *fp_A, 
        const int *ind, const int len){


    const int blockSize = 128;
 

    half * tmp = const_cast<half*>(A);
    dim3 numBlocks(len);        
    FindOutliersAndSetToZeros_kernel_<<<numBlocks, blockSize, 1024>>>(
            ind,
            tmp,
            fp_A,
            M,
            N,
            len
        );

}
void test(const int m, const int n, const int k) {

    hipblasHandle_t handle;
    hipblasStatus_t status = hipblasCreate(&handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
    std::cerr << "Failed to create cuBLAS handle" << std::endl;
    exit(0);
    }
    hipEvent_t start, stop;
    float elapsedTime;
    int loop = 300;


    cutlass::half_t   *C_cublas_ptr, *A_cublas_ptr, *B_cublas_ptr , *B_quant_ptr, *A_quant_ptr;
    hipMalloc(&C_cublas_ptr, sizeof(cutlass::half_t) * m * n);
    hipMalloc(&A_cublas_ptr, sizeof(cutlass::half_t) * m * k);
    hipMalloc(&B_cublas_ptr, sizeof(cutlass::half_t) * n * k);
    hipMalloc(&A_quant_ptr, sizeof(cutlass::half_t) * m * k);
    hipMalloc(&B_quant_ptr, sizeof(cutlass::half_t) * n * k);

    cutlass::half_t * hostA_cublas_ptr = (cutlass::half_t *)malloc(sizeof(cutlass::half_t) * m * k);
    cutlass::half_t * hostB_cublas_ptr = (cutlass::half_t *)malloc(sizeof(cutlass::half_t) * n * k);
    cutlass::half_t * hostC_cublas_ptr = (cutlass::half_t *)malloc(sizeof(cutlass::half_t) * m * n);
    gen_rand_data<cutlass::half_t>(hostA_cublas_ptr, m * k);
    gen_rand_data<cutlass::half_t>(hostB_cublas_ptr, n * k);



    hipMemcpy(A_cublas_ptr, hostA_cublas_ptr, sizeof(cutlass::half_t) * m * k, hipMemcpyHostToDevice);
    hipMemcpy(B_cublas_ptr, hostB_cublas_ptr, sizeof(cutlass::half_t) * n * k, hipMemcpyHostToDevice);





     //---------------------------------------------------------------------------------------------------
    cutlass::half_t   *C_half_out_ptr, *tmp_ptr;
    int8_t  *int8_out_A, *int8_out_B;
    int8_t  *int8_debug_ptr  = (int8_t *)malloc(sizeof(int8_t) * k * n * 4);
    cutlass::half_t   *scale_a, *scale_b;
    hipMalloc(&C_half_out_ptr, sizeof(cutlass::half_t) * m * n);
    hipMalloc(&tmp_ptr, sizeof(cutlass::half_t) * m * n);
    hipMalloc(&int8_out_A, sizeof(int8_t) * m * k);
    hipMalloc(&int8_out_B, sizeof(int8_t) * n * k);

    hipMalloc(&scale_a, sizeof(cutlass::half_t) * m);
    hipMalloc(&scale_b, sizeof(cutlass::half_t) * n);

    const int num_ind = 128;
    int8quant_(m, k, A_cublas_ptr, int8_out_A, scale_a);
    int8quant_(n, k, B_cublas_ptr, int8_out_B, scale_b);



    hipDeviceSynchronize();


    // printf("half input---------------\n");
    // hipMemcpy(hostC_cublas_ptr, A_cublas_ptr, sizeof(cutlass::half_t) * n * k, hipMemcpyDeviceToHost);
    // for (int i = 0 ; i < 20; ++i) {
    //     float tmp = (float) hostC_cublas_ptr[i];
    //     printf("%.4f\t", tmp);
    // }
    // printf("scale a output---------------\n");
    // hipMemcpy(hostC_cublas_ptr, scale_a, sizeof(cutlass::half_t) * m , hipMemcpyDeviceToHost);
    // for (int i = 0 ; i < 20; ++i) {
    //     float tmp = (float) hostC_cublas_ptr[i];
    //     printf("%.4f\t", tmp);
    // }

    // return ;
    // hipMemcpy(hostC_cublas_ptr, scale_b, sizeof(cutlass::half_t) * n, hipMemcpyDeviceToHost);
    // for (int i = 0 ; i < 20; ++i) {
    //     float tmp = (float) hostC_cublas_ptr[i];
    //     printf("%.4f\t", tmp);
    // }
    // printf("b scales---------------\n");
    // hipMemcpy(int8_debug_ptr, int8_out_A, sizeof(int8_t) * m * k, hipMemcpyDeviceToHost);
    // for (int i = 0 ; i < 20; ++i) {
    //     float tmp = (float) int8_debug_ptr[i];
    //     printf("%.4f\t", tmp);
    // }
    // printf("int 8 out---------------\n");
    // hipMemcpy(int8_debug_ptr, int8_out_B, sizeof(int8_t) * n * k, hipMemcpyDeviceToHost);
    // for (int i = 0 ; i < 20; ++i) {
    //     float tmp = (float) int8_debug_ptr[i];
    //     printf("%.4f\t", tmp);
    // }
     
    // hipDeviceSynchronize();
    // return;



    

    // cutlass::half_t   *fp_activation, *fp_weight;
    // hipMalloc(&fp_activation, sizeof(cutlass::half_t) * m * num_ind);
    // hipMalloc(&fp_weight, sizeof(cutlass::half_t) * n * num_ind);


    // int *ind_cpu =   (int *)malloc(sizeof(int) * num_ind);
    // for (int i = 0 ; i < num_ind; ++i) ind_cpu[i] = i;
    // int   *ind ;
    // hipMalloc(&ind, sizeof(int) *  num_ind);
    // hipMemcpy(ind, ind_cpu, sizeof(ind) * num_ind, hipMemcpyHostToDevice);

    // ExtractOutliersAndSetToZeros_(m, k, ( half *)A_quant_ptr, ( half *)fp_activation, ind, num_ind);
    // ExtractOutliersAndSetToZeros_(n, k, ( half *)B_quant_ptr, ( half *)fp_weight, ind, num_ind);
    // int8quant_(m, k, A_quant_ptr, int8_out_A, scale_a);
    // int8quant_(n, k, B_quant_ptr, int8_out_B, scale_b);

    // hipEventCreate(&start);
    // hipEventCreate(&stop);
    // // cublas
    // for (int i = 0; i < 10; ++i){
    //     gemmfp16_((half *)fp_activation, (half *)fp_weight, (half *)C_half_out_ptr, m, n, num_ind, handle);
    //     int8FusedDequantizeCUDA_(int8_out_A, int8_out_B, 
    //                         scale_a,
    //                         scale_b, 
    //                         C_half_out_ptr, 
    //                         C_half_out_ptr, 
    //                         m, n, k, 
    //                         reinterpret_cast<char*>(tmp_ptr));

    // }
 
    // hipEventRecord(start, 0);
    // for (int i = 0; i < loop; ++i){
    //     gemmfp16_((half *)fp_activation, (half *)fp_weight, (half *)C_half_out_ptr, m, n, num_ind, handle);
    //     int8FusedDequantizeCUDA_(int8_out_A, int8_out_B, 
    //                         scale_a,
    //                         scale_b, 
    //                         C_half_out_ptr, 
    //                         C_half_out_ptr, 
    //                         m, n, k, 
    //                         reinterpret_cast<char*>(tmp_ptr));

    // }
    // hipEventRecord(stop, 0);
    // hipEventSynchronize(stop);
    // hipEventElapsedTime(&elapsedTime, start, stop);
    // printf("int8gemm + outliers FLOPs: %5.3fT\n", (2*float(n)*float(m)*float(k)/elapsedTime)/1e9 * loop);
    // hipDeviceSynchronize();
    // ---------------------------------------------------------------------


    
    cutlass::half_t alpha = cutlass::half_t(1.0f);
    cutlass::half_t beta = cutlass::half_t(0.0f);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // cublas
    for (int i = 0; i < 10; ++i)
        hipblasHgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k,
                    &reinterpret_cast<const __half&>(alpha), (__half*)B_cublas_ptr, k, (__half*)A_cublas_ptr,
                     k, &reinterpret_cast<const __half&>(beta), (__half*)C_cublas_ptr, n);
 
    hipEventRecord(start, 0);
    for (int i = 0; i < loop; ++i)
        hipblasHgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, m, k,
                    &reinterpret_cast<const __half&>(alpha), (__half*)B_cublas_ptr, k, (__half*)A_cublas_ptr,
                     k, &reinterpret_cast<const __half&>(beta), (__half*)C_cublas_ptr, n);
        

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("%5.3fT,\t", (2*float(n)*float(m)*float(k)/elapsedTime)/1e9 * loop);
    hipDeviceSynchronize();


    hipMemcpy(hostC_cublas_ptr, C_cublas_ptr, sizeof(cutlass::half_t) * m * n, hipMemcpyDeviceToHost);
    
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // cublas
    for (int i = 0; i < 10; ++i)
        int8FusedDequantizeCUDA_(int8_out_A, int8_out_B, 
                            scale_a,
                            scale_b, 
                            C_half_out_ptr, 
                            C_half_out_ptr, 
                            m, n, k, 
                            reinterpret_cast<char*>(tmp_ptr));
 
    hipEventRecord(start, 0);
    for (int i = 0; i < loop; ++i)
        int8FusedDequantizeCUDA_(int8_out_A, int8_out_B, 
                            scale_a,
                            scale_b, 
                            C_half_out_ptr, 
                            C_half_out_ptr, 
                            m, n, k, 
                            reinterpret_cast<char*>(tmp_ptr));
        

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("%5.3fT,\t", (2*float(n)*float(m)*float(k)/elapsedTime)/1e9 * loop);
    hipDeviceSynchronize();

   

    hipMemcpy(A_quant_ptr, hostA_cublas_ptr, sizeof(cutlass::half_t) * m * k, hipMemcpyHostToDevice);
    hipMemcpy(B_quant_ptr, hostB_cublas_ptr, sizeof(cutlass::half_t) * n * k, hipMemcpyHostToDevice);

    cutlass::half_t   *fp_activation, *fp_weight;
    hipMalloc(&fp_activation, sizeof(cutlass::half_t) * m * num_ind);
    hipMalloc(&fp_weight, sizeof(cutlass::half_t) * n * num_ind);


    int *ind_cpu =   (int *)malloc(sizeof(int) * num_ind);
    for (int i = 0 ; i < num_ind; ++i) ind_cpu[i] = i;
    int   *ind ;
    hipMalloc(&ind, sizeof(int) *  num_ind);
    hipMemcpy(ind, ind_cpu, sizeof(int) * num_ind, hipMemcpyHostToDevice);

    ExtractOutliersAndSetToZeros_(m, k, ( half *)A_quant_ptr, ( half *)fp_activation, ind, num_ind);
    ExtractOutliersAndSetToZeros_(n, k, ( half *)B_quant_ptr, ( half *)fp_weight, ind, num_ind);
    int8quant_(m, k, A_quant_ptr, int8_out_A, scale_a);
    int8quant_(n, k, B_quant_ptr, int8_out_B, scale_b);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int i = 0; i < 10; ++i){
        gemmfp16_((half *)fp_activation, (half *)fp_weight, (half *)C_half_out_ptr, m, n, num_ind, handle);
        int8FusedDequantizeCUDA_(int8_out_A, int8_out_B, 
                            scale_a,
                            scale_b, 
                            C_half_out_ptr, 
                            C_half_out_ptr, 
                            m, n, k, 
                            reinterpret_cast<char*>(tmp_ptr));

    }
 
    hipEventRecord(start, 0);
    for (int i = 0; i < loop; ++i){
        gemmfp16_((half *)fp_activation, (half *)fp_weight, (half *)C_half_out_ptr, m, n, num_ind, handle);
        int8FusedDequantizeCUDA_(int8_out_A, int8_out_B, 
                            scale_a,
                            scale_b, 
                            C_half_out_ptr, 
                            C_half_out_ptr, 
                            m, n, k, 
                            reinterpret_cast<char*>(tmp_ptr));

    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("%5.3fT\n", (2*float(n)*float(m)*float(k)/elapsedTime)/1e9 * loop);
    hipDeviceSynchronize();
    //---------------------------------------------------------------------

    if (m == 4096){
        printf("\n");
        for (int i = 0 ; i < 20; ++i) {
            float tmp = (float) hostC_cublas_ptr[i];
            printf("%.4f\t", tmp);
        }
        printf("---------------\n");


        hipMemset(C_half_out_ptr, 0, sizeof(half) * m * n);
        gemmfp16_((half *)fp_activation, (half *)fp_weight, (half *)C_half_out_ptr, m, n, num_ind, handle);
        int8FusedDequantizeCUDA_(int8_out_A, int8_out_B, 
                            scale_a,
                            scale_b, 
                            C_half_out_ptr, 
                            C_half_out_ptr, 
                            m, n, k, 
                            reinterpret_cast<char*>(tmp_ptr));
        hipMemcpy(hostC_cublas_ptr, C_half_out_ptr, sizeof(cutlass::half_t) * m * n, hipMemcpyDeviceToHost);
        for (int i = 0 ; i < 20; ++i) {
            float tmp = (float) hostC_cublas_ptr[i];
            printf("%.4f\t", tmp);
        }
        printf("---------------\n");

    }

}
 

int main() {
    srand(10086);

    printf("cublas INT8,\tfused INT8,\tmixed INT8\n");
    const int n = 4096;
    const int k = 4096;
    
    // test(128,n,k);
    // test(256,n,k);
    test(512,n,k);
    test(1024,n,k);
    test(2048,n,k);
    test(4096,n,k);
    // test(8192,n,k);

    return 0;
}